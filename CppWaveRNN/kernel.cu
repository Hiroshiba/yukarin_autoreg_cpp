#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#include <>

#include "CppWaveRNN.h"

// CUDA Runtime error messages
#ifdef __DRIVER_TYPES_H__
static const char *_cudaGetErrorEnum(hipError_t error)
{
	return hipGetErrorName(error);
}
#endif

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";

	case HIPBLAS_STATUS_NOT_SUPPORTED:
		return "HIPBLAS_STATUS_NOT_SUPPORTED";

	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
	}

	return "<unknown>";
}
#endif

#ifdef CUDNN_H_
// cuDNN API errors
static const char *_cudaGetErrorEnum(hipdnnStatus_t error)
{
	return hipdnnGetErrorString(error);
}
#endif

template <typename T>
void check(T result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
				static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}

#ifdef __DRIVER_TYPES_H__
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
#endif

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

template <typename T>
T *cudaMallocUtil(int size, T *h = NULL)
{
	T *x;
	checkCudaErrors(hipMalloc(&x, size * sizeof(T)));
	if (h != NULL)
	{
		checkCudaErrors(hipMemcpy(x, h, size * sizeof(T), hipMemcpyHostToDevice));
	}
	return x;
}

template <typename T>
struct ndarray
{
	T *device;
	T *host;
	int shape1 = 1;
	int shape2 = 1;
	int shape3 = 1;
	ndarray(T *h = NULL) : host(h)
	{
	}
	ndarray(int s1, T *h = NULL) : shape1(s1), host(h)
	{
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, T *h = NULL) : shape1(s1), shape2(s2), host(h)
	{
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, int s3, T *h = NULL) : shape1(s1), shape2(s2), shape3(s3), host(h)
	{
		device = cudaMallocUtil<T>(size(), host);
	}

	int size()
	{
		return shape1 * shape2 * shape3;
	}
};

__global__ void concat(float *xl, int *x, float *l, float *x_embedder_W, int batch_size, int local_size, int embedding_size)
{
	int feature_size = embedding_size + local_size;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * feature_size)
	{
		return;
	}

	int i_batch = i / feature_size;
	int i_feature = i % feature_size;

	if (i_feature < embedding_size)
	{
		// embedding
		int i_x = x[i_batch];
		int i_embedding = i_x * embedding_size + i_feature;
		xl[i] = x_embedder_W[i_embedding];
	}
	else
	{
		// local
		int i_local = i_batch * local_size + (i_feature - embedding_size);
		xl[i] = l[i_local];
	}
}

__global__ void gruElementWise(
	float *hidden,
	float *W,
	float *U,
	int batch_size,
	int hidden_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * hidden_size)
		return;

	float W_r_x = W[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float U_r_h = U[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float r = tanh((W_r_x + U_r_h) * 0.5f) * 0.5f + 0.5f;

	float W_z_x = W[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float U_z_h = U[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float z = tanh((W_z_x + U_z_h) * 0.5f) * 0.5f + 0.5f;

	float W_x = W[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float U_x = U[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float h_bar = tanh(W_x + r * U_x);

	hidden[i] = z * hidden[i] + (1.f - z) * h_bar;
}

__global__ void relu(float *x, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	if (x[i] < 0)
		x[i] = 0;
}

__global__ void initRandomState(hiprandState *state, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	hiprand_init(i, 0, 0, &state[i]);
}

__global__ void floatToDouble(float *src, double *dst, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	dst[i] = (double) src[i];
}

__global__ void addGumbel(double *x, hiprandState *state, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	x[i] += -log(-log(hiprand_uniform_double(&state[i])));
}

__global__ void pairToKey(int *x, hipcub::KeyValuePair<int, double> *pair, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	x[i] = pair[i].key;
}

int *h_pinned_x;
float *h_pinned_l_array;
float *h_pinned_hidden;
int *h_pinned_output;

ndarray<int> *x;
ndarray<float> *l_array;
ndarray<float> *hidden;

ndarray<float> *x_embedder_W;
ndarray<float> *gru_xw;
ndarray<float> *gru_xb;
ndarray<float> *gru_hw;
ndarray<float> *gru_hb;
ndarray<float> *O1_W;
ndarray<float> *O1_b;
ndarray<float> *O2_W;
ndarray<float> *O2_b;

ndarray<float> *gru_xb_b;
ndarray<float> *gru_hb_b;
ndarray<float> *O1_b_b;
ndarray<float> *O2_b_b;

ndarray<float> *xl;
ndarray<float> *w_gru_x;
ndarray<float> *w_gru_h;
ndarray<float> *w_out_x1;
ndarray<float> *w_out_x2;
ndarray<double> *w_dist;
ndarray<hipcub::KeyValuePair<int, double>> *w_sampled;

ndarray<hiprandState> *gumbel_random_state;

ndarray<int> *argmax_offset;
size_t argmax_storage_bytes;
ndarray<char> *argmax_storage;

hipStream_t stream;
hipStream_t biasCopyStream;
hipStream_t hiddenStream;
hipStream_t outputCopyStream;
hipblasHandle_t cublasHandle;
hipblasHandle_t cublasHiddenHandle;
hipdnnHandle_t cudnnHandle;

hipEvent_t copyHiddenDone;
hipEvent_t elementWiseDone, gemmO2Done, argmaxDone;
hipEvent_t copyGruXbDone, copyGruHbDone, copyO1bDone, copyO2bDone;
hipEvent_t gemmO1Done;
hipEvent_t gemmGruHDone;
hipEvent_t outputCopyDone;
hipEvent_t toKeyDone;
hipdnnTensorDescriptor_t softmaxDesc;

int g_graph_length;
int g_local_size;
int g_hidden_size;
int g_embedding_size;

void initialize(
	int graph_length,
	int max_batch_size,
	int local_size,
	int hidden_size,
	int embedding_size,
	int linear_hidden_size,
	int output_size,
	float *h_x_embedder_W,
	float *h_gru_xw,
	float *h_gru_xb,
	float *h_gru_hw,
	float *h_gru_hb,
	float *h_O1_W,
	float *h_O1_b,
	float *h_O2_W,
	float *h_O2_b)
{
	// initialize
	std::cout << "initialize" << std::endl;
	checkCudaErrors(hipHostAlloc(&h_pinned_x, max_batch_size * sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc(&h_pinned_l_array, graph_length * max_batch_size * local_size * sizeof(float), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc(&h_pinned_hidden, max_batch_size * hidden_size * sizeof(float), hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc(&h_pinned_output, graph_length * max_batch_size * sizeof(int), hipHostMallocDefault));

	x = new ndarray<int>(max_batch_size, h_pinned_x);
	l_array = new ndarray<float>(graph_length, max_batch_size, local_size, h_pinned_l_array);
	hidden = new ndarray<float>(max_batch_size, hidden_size, h_pinned_hidden);

	x_embedder_W = new ndarray<float>(output_size, embedding_size, h_x_embedder_W);
	gru_xw = new ndarray<float>(embedding_size + local_size, hidden_size * 3, h_gru_xw);
	gru_xb = new ndarray<float>(hidden_size * 3, h_gru_xb);
	gru_hw = new ndarray<float>(hidden_size, hidden_size * 3, h_gru_hw);
	gru_hb = new ndarray<float>(hidden_size * 3, h_gru_hb);
	O1_W = new ndarray<float>(hidden_size, linear_hidden_size, h_O1_W);
	O1_b = new ndarray<float>(linear_hidden_size, h_O1_b);
	O2_W = new ndarray<float>(linear_hidden_size, output_size, h_O2_W);
	O2_b = new ndarray<float>(output_size, h_O2_b);

	gru_xb_b = new ndarray<float>(max_batch_size, gru_xb->shape1);
	gru_hb_b = new ndarray<float>(max_batch_size, gru_hb->shape1);
	O1_b_b = new ndarray<float>(max_batch_size, O1_b->shape1);
	O2_b_b = new ndarray<float>(max_batch_size, O2_b->shape1);

	xl = new ndarray<float>(max_batch_size, embedding_size + local_size);
	w_gru_x = new ndarray<float>(max_batch_size, hidden_size * 3);
	w_gru_h = new ndarray<float>(max_batch_size, hidden_size * 3);
	w_out_x1 = new ndarray<float>(max_batch_size, linear_hidden_size);
	w_out_x2 = new ndarray<float>(max_batch_size, output_size);
	w_dist = new ndarray<double>(max_batch_size, output_size);
	w_sampled = new ndarray<hipcub::KeyValuePair<int, double>>(max_batch_size);

	gumbel_random_state = new ndarray<hiprandState>(max_batch_size, output_size);

	// create context
	std::cout << "create context" << std::endl;

	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamDefault));
	checkCudaErrors(hipStreamCreateWithFlags(&biasCopyStream, hipStreamDefault));
	checkCudaErrors(hipStreamCreateWithFlags(&hiddenStream, hipStreamDefault));
	checkCudaErrors(hipStreamCreateWithFlags(&outputCopyStream, hipStreamDefault));
	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipblasSetStream(cublasHandle, stream));
	checkCudaErrors(hipblasCreate(&cublasHiddenHandle));
	checkCudaErrors(hipblasSetStream(cublasHiddenHandle, hiddenStream));

	for (int i = 0; i < max_batch_size; i++)
	{
		// broadcast
		checkCudaErrors(hipMemcpyAsync(&gru_xb_b->device[i * gru_xb_b->shape2], gru_xb->device, gru_xb_b->shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(&gru_hb_b->device[i * gru_hb_b->shape2], gru_hb->device, gru_hb_b->shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(&O1_b_b->device[i * O1_b_b->shape2], O1_b->device, O1_b_b->shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(&O2_b_b->device[i * O2_b_b->shape2], O2_b->device, O2_b_b->shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
	}

	checkCudaErrors(hipdnnCreate(&cudnnHandle));
	checkCudaErrors(hipdnnSetStream(cudnnHandle, stream));

	initRandomState KERNEL_ARGS4(dim3(gumbel_random_state->size() / 512 + 1), dim3(512), 0, stream)(
		gumbel_random_state->device, // hiprandState *state,
		gumbel_random_state->size()	 // int size
	);

	int *h_argmax_offset = (int *)malloc((w_out_x2->shape1 + 1) * sizeof(int));
	for (int i = 0; i < w_out_x2->shape1 + 1; i++)
	{
		h_argmax_offset[i] = i * w_out_x2->shape2;
	}
	argmax_offset = new ndarray<int>(w_out_x2->shape1 + 1, h_argmax_offset);

	argmax_storage_bytes = 0;
	checkCudaErrors(hipcub::DeviceSegmentedReduce::ArgMax(
		NULL,					   // void *d_temp_storage
		argmax_storage_bytes,	   // size_t &temp_storage_bytes
		w_out_x2->device,		   // InputIteratorT d_in
		w_sampled->device,		   // OutputIteratorT d_out
		w_out_x2->shape1,		   // int num_segments
		argmax_offset->device,	   // OffsetIteratorT d_begin_offsets
		argmax_offset->device + 1, // OffsetIteratorT d_end_offsets
		0,						   // hipStream_t stream
		true					   // bool debug_synchronous
		));
	argmax_storage = new ndarray<char>((int)argmax_storage_bytes);

	// graph
	checkCudaErrors(hipEventCreateWithFlags(&copyHiddenDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&elementWiseDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&gemmO2Done, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&argmaxDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&copyGruXbDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&copyGruHbDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&copyO1bDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&copyO2bDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&gemmO1Done, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&gemmGruHDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&outputCopyDone, hipEventDisableTiming));
	checkCudaErrors(hipEventCreateWithFlags(&toKeyDone, hipEventDisableTiming));

	checkCudaErrors(hipdnnCreateTensorDescriptor(&softmaxDesc));
	checkCudaErrors(hipdnnSetTensor4dDescriptor(
		softmaxDesc,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_DOUBLE,
		max_batch_size,
		w_out_x2->shape2,
		1,
		1));

	// copy to global
	g_graph_length = graph_length;
	g_local_size = local_size;
	g_hidden_size = hidden_size;
	g_embedding_size = embedding_size;
}

void destoroy()
{
	checkCudaErrors(hipEventDestroy(elementWiseDone));
	checkCudaErrors(hipEventDestroy(gemmO2Done));
	checkCudaErrors(hipEventDestroy(argmaxDone));
	checkCudaErrors(hipEventDestroy(copyGruXbDone));
	checkCudaErrors(hipEventDestroy(copyGruHbDone));
	checkCudaErrors(hipEventDestroy(copyO1bDone));
	checkCudaErrors(hipEventDestroy(copyO2bDone));
	checkCudaErrors(hipEventDestroy(gemmO1Done));
	checkCudaErrors(hipEventDestroy(gemmGruHDone));
	checkCudaErrors(hipEventDestroy(outputCopyDone));
	checkCudaErrors(hipEventDestroy(toKeyDone));

	checkCudaErrors(hipStreamDestroy(biasCopyStream));
	checkCudaErrors(hipStreamDestroy(hiddenStream));
	checkCudaErrors(hipStreamDestroy(outputCopyStream));
}

void inference(
	int batch_size,
	int length,
	int *h_output,
	int *h_x,
	float *h_l_array,
	float *h_hidden)
{
	// std::chrono::system_clock::time_point start, end;
	// start = std::chrono::system_clock::now();

	checkCudaErrors(hipMemcpyAsync(x->host, h_x, batch_size * sizeof(int), hipMemcpyHostToHost, stream));
	checkCudaErrors(hipMemcpyAsync(hidden->host, h_hidden, batch_size * hidden->shape2 * sizeof(float), hipMemcpyHostToHost, stream));

	int l_size = batch_size * l_array->shape3;

	int max_batch_size = l_array->shape2;
	int g_l_size = l_array->shape2 * l_array->shape3;

	int now_length = 0;
	while (now_length < length)
	{
		// re-zero
		checkCudaErrors(hipMemsetAsync(l_array->host, 0, l_array->size(), stream));

		// choice graph length
		int next_length;
		if (length - now_length >= g_graph_length)
		{
			next_length = g_graph_length;
		}
		else
		{
			next_length = length - now_length;
		}

		// forward
		for (int i = 0; i < next_length; i++)
		{
			checkCudaErrors(hipMemcpyAsync(&l_array->host[i * g_l_size], &h_l_array[(now_length + i) * l_size], l_size * sizeof(float), hipMemcpyHostToHost, stream));
		}

		// std::cout << "start" << std::endl;

		checkCudaErrors(hipMemcpyAsync(x->device, x->host, max_batch_size * sizeof(int), hipMemcpyHostToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(l_array->device, l_array->host, l_array->shape1 * max_batch_size * l_array->shape3 * sizeof(float), hipMemcpyHostToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(hidden->device, hidden->host, max_batch_size * hidden->shape2 * sizeof(float), hipMemcpyHostToDevice, stream));
		hipEventRecord(copyHiddenDone, stream);
		hipStreamWaitEvent(hiddenStream, copyHiddenDone, 0);

		for (int i_local = 0; i_local < g_graph_length; i_local++)
		{
			// concat
			concat KERNEL_ARGS4(dim3(512), dim3(max_batch_size * xl->shape2 / 512 + 1), 0, stream)(
				xl->device,														// float* xl,
				x->device,														// int* x,
				&l_array->device[i_local * (max_batch_size * l_array->shape3)], // float* l,
				x_embedder_W->device,											// float* x_embedder_W,
				max_batch_size,													// int batch_size,
				g_local_size,													// int local_size,
				g_embedding_size												// int embedding_size
			);

			// gru_x = prev_xl.dot(gru_xw) + gru_xb
			hipStreamWaitEvent(biasCopyStream, elementWiseDone, 0);
			checkCudaErrors(hipMemcpyAsync(w_gru_x->device, gru_xb_b->device, max_batch_size * w_gru_x->shape2 * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
			hipEventRecord(copyGruXbDone, biasCopyStream);

			float gemmAlpha = 1, gemmBeta = 1;
			hipStreamWaitEvent(stream, copyGruXbDone, 0);
			checkCudaErrors(hipblasSgemm(
				cublasHandle,	 // hipblasHandle_t handle,
				HIPBLAS_OP_N,	 // hipblasOperation_t transa,
				HIPBLAS_OP_N,	 // hipblasOperation_t transb,
				gru_xw->shape2,	 // int m,
				max_batch_size,	 // int n,
				gru_xw->shape1,	 // int k,
				&gemmAlpha,		 // const float *alpha, /* host or device pointer */
				gru_xw->device,	 // const float *A,
				gru_xw->shape2,	 // int lda,
				xl->device,		 // const float *B,
				xl->shape2,		 // int ldb,
				&gemmBeta,		 // const float *beta, /* host or device pointer */
				w_gru_x->device, // float *C,
				w_gru_x->shape2	 // int ldc
				));

			// gru_h = hidden.dot(gru_hw) + gru_hb
			checkCudaErrors(hipStreamWaitEvent(biasCopyStream, elementWiseDone, 0));
			checkCudaErrors(hipMemcpyAsync(w_gru_h->device, gru_hb_b->device, max_batch_size * w_gru_h->shape2 * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
			checkCudaErrors(hipEventRecord(copyGruHbDone, biasCopyStream));

			checkCudaErrors(hipStreamWaitEvent(hiddenStream, copyGruHbDone, 0));
			checkCudaErrors(hipStreamWaitEvent(hiddenStream, gemmO1Done, 0));
			checkCudaErrors(hipblasSgemm(
				cublasHiddenHandle, // hipblasHandle_t handle,
				HIPBLAS_OP_N,		// hipblasOperation_t transa,
				HIPBLAS_OP_N,		// hipblasOperation_t transb,
				gru_hw->shape2,		// int m,
				max_batch_size,		// int n,
				gru_hw->shape1,		// int k,
				&gemmAlpha,			// const float *alpha, /* host or device pointer */
				gru_hw->device,		// const float *A,
				gru_hw->shape2,		// int lda,
				hidden->device,		// const float *B,
				hidden->shape2,		// int ldb,
				&gemmBeta,			// const float *beta, /* host or device pointer */
				w_gru_h->device,	// float *C,
				w_gru_h->shape2		// int ldc
				));
			checkCudaErrors(hipEventRecord(gemmGruHDone, hiddenStream));

			// gruElementWise
			hipStreamWaitEvent(stream, gemmGruHDone, 0);
			gruElementWise KERNEL_ARGS4(dim3(max_batch_size * hidden->shape2 / 512 + 1), dim3(512), 0, stream)(
				hidden->device,	 // float* hidden
				w_gru_x->device, // float* W
				w_gru_h->device, // float* U
				max_batch_size,	 // int batch_size
				g_hidden_size	 // int hidden_size
			);
			hipEventRecord(elementWiseDone, stream);

			// out_x = hidden.dot(O1_W) + O1_b
			hipStreamWaitEvent(biasCopyStream, gemmO2Done, 0);
			checkCudaErrors(hipMemcpyAsync(w_out_x1->device, O1_b_b->device, max_batch_size * w_out_x1->shape2 * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
			hipEventRecord(copyO1bDone, biasCopyStream);

			hipStreamWaitEvent(stream, copyO1bDone, 0);
			checkCudaErrors(hipblasSgemm(
				cublasHandle,	  // hipblasHandle_t handle,
				HIPBLAS_OP_N,	  // hipblasOperation_t transa,
				HIPBLAS_OP_N,	  // hipblasOperation_t transb,
				O1_W->shape2,	  // int m,
				max_batch_size,	  // int n,
				O1_W->shape1,	  // int k,
				&gemmAlpha,		  // const float *alpha, /* host or device pointer */
				O1_W->device,	  // const float *A,
				O1_W->shape2,	  // int lda,
				hidden->device,	  // const float *B,
				hidden->shape2,	  // int ldb,
				&gemmBeta,		  // const float *beta, /* host or device pointer */
				w_out_x1->device, // float *C,
				w_out_x1->shape2  // int ldc
				));
			hipEventRecord(gemmO1Done, stream);

			// relu
			relu KERNEL_ARGS4(dim3(max_batch_size * w_out_x1->shape2 / 512 + 1), dim3(512), 0, stream)(
				w_out_x1->device,				  // float* x
				max_batch_size * w_out_x1->shape2 // int size
			);

			// out_x = out_x.dot(O2_W) + O2_b
			hipStreamWaitEvent(biasCopyStream, argmaxDone, 0);
			checkCudaErrors(hipMemcpyAsync(w_out_x2->device, O2_b_b->device, max_batch_size * w_out_x2->shape2 * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
			hipEventRecord(copyO2bDone, biasCopyStream);

			hipStreamWaitEvent(stream, copyO2bDone, 0);
			checkCudaErrors(hipblasSgemm(
				cublasHandle,	  // hipblasHandle_t handle,
				HIPBLAS_OP_N,	  // hipblasOperation_t transa,
				HIPBLAS_OP_N,	  // hipblasOperation_t transb,
				O2_W->shape2,	  // int m,
				max_batch_size,	  // int n,
				O2_W->shape1,	  // int k,
				&gemmAlpha,		  // const float *alpha, /* host or device pointer */
				O2_W->device,	  // const float *A,
				O2_W->shape2,	  // int lda,
				w_out_x1->device, // const float *B,
				w_out_x1->shape2, // int ldb,
				&gemmBeta,		  // const float *beta, /* host or device pointer */
				w_out_x2->device, // float *C,
				w_out_x2->shape2  // int ldc
				));
			hipEventRecord(gemmO2Done, stream);

			floatToDouble KERNEL_ARGS4(dim3(max_batch_size * w_out_x2->shape2 / 512 + 1), dim3(512), 0, stream)(
				w_out_x2->device,					// float *src
				w_dist->device,						// double *dst
				max_batch_size * w_out_x2->shape2	// int size
			);

			// softmax
			double softmaxAlpha = 1, softmaxBeta = 0;
			checkCudaErrors(hipdnnSoftmaxForward(
				cudnnHandle,				// hipdnnHandle_t
				HIPDNN_SOFTMAX_LOG,			// hipdnnSoftmaxAlgorithm_t
				HIPDNN_SOFTMAX_MODE_CHANNEL, // hipdnnSoftmaxMode_t
				&softmaxAlpha,				// const void
				softmaxDesc,				// const hipdnnTensorDescriptor_t
				w_dist->device,				// const void
				&softmaxBeta,				// const void
				softmaxDesc,				// const hipdnnTensorDescriptor_t
				w_dist->device				// void
				));

			// sampling
			addGumbel KERNEL_ARGS4(dim3(max_batch_size * w_dist->shape2 / 512 + 1), dim3(512), 0, stream)(
				w_dist->device,				  // double *x
				gumbel_random_state->device,  // hiprandState *state
				max_batch_size * w_dist->shape2 // int size
			);

			checkCudaErrors(hipcub::DeviceSegmentedReduce::ArgMax(
				argmax_storage->device,	   // void *d_temp_storage
				argmax_storage_bytes,	   // size_t &temp_storage_bytes
				w_dist->device,			   // InputIteratorT d_in
				w_sampled->device,		   // OutputIteratorT d_out
				max_batch_size,			   // int num_segments
				argmax_offset->device,	   // OffsetIteratorT d_begin_offsets
				argmax_offset->device + 1, // OffsetIteratorT d_end_offsets
				stream,					   // hipStream_t stream
				false					   // bool debug_synchronous
				));
			hipEventRecord(argmaxDone, stream);

			hipStreamWaitEvent(stream, outputCopyDone, 0);
			pairToKey KERNEL_ARGS4(dim3(max_batch_size / 512 + 1), dim3(512), 0, stream)(
				x->device,		   // int *x
				w_sampled->device, // hipcub::KeyValuePair<int, double>* pair
				max_batch_size	   // int size
			);
			hipEventRecord(toKeyDone, stream);

			hipStreamWaitEvent(outputCopyStream, toKeyDone, 0);
			hipMemcpyAsync(&h_pinned_output[i_local * max_batch_size], x->device, max_batch_size * sizeof(int), hipMemcpyDeviceToHost, outputCopyStream);
			hipEventRecord(outputCopyDone, outputCopyStream);
		}

		hipMemcpyAsync(x->host, x->device, max_batch_size * sizeof(int), hipMemcpyDeviceToHost, stream);
		hipMemcpyAsync(hidden->host, hidden->device, max_batch_size * hidden->shape2 * sizeof(float), hipMemcpyDeviceToHost, stream);

		hipStreamWaitEvent(stream, outputCopyDone, 0);

		// std::cout << "done" << std::endl;

		for (int i = 0; i < next_length; i++)
		{
			checkCudaErrors(hipMemcpyAsync(&h_output[(now_length + i) * batch_size], &h_pinned_output[i * max_batch_size], batch_size * sizeof(int), hipMemcpyHostToHost, stream));
		}

		// next loop
		now_length += next_length;
	}

	checkCudaErrors(hipMemcpyAsync(h_x, x->host, batch_size * sizeof(int), hipMemcpyHostToHost, stream));
	checkCudaErrors(hipMemcpyAsync(h_hidden, hidden->host, batch_size * hidden->shape2 * sizeof(float), hipMemcpyHostToHost, stream));

	checkCudaErrors(hipStreamSynchronize(stream));

	// end = std::chrono::system_clock::now();

	// double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000 / 1000;
	// printf("time %lf[s]\n", time);
}
