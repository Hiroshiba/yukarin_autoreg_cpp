#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>
#include <cub.cuh>
#include <util_type.cuh>
#include <>

#include "CppWaveRNN.h"


#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


template<typename T>
T* cudaMallocUtil(int size, T* h = NULL) {
	T* x;
	cudaErrorCheckUtil(hipMalloc(&x, size * sizeof(T)));
	if (h != NULL) {
		cudaErrorCheckUtil(hipMemcpy(x, h, size * sizeof(T), hipMemcpyHostToDevice));
	}
	return x;
}


template<typename T>
struct ndarray {
	T* device;
	T* host;
	int shape1 = 1;
	int shape2 = 1;
	int shape3 = 1;
	ndarray(int s1, T* h = NULL) : shape1(s1), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, T* h = NULL) : shape1(s1), shape2(s2), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, int s3, T* h = NULL) : shape1(s1), shape2(s2), shape3(s3), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}

	int size() {
		return shape1 * shape2 * shape3;
	}
};


__global__ void concat(float* xl, int* x, float* l, float* x_embedder_W, int batch_size, int local_size, int embedding_size)
{
	int feature_size = embedding_size + local_size;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * feature_size) { return; }

	int i_batch = i / feature_size;
	int i_feature = i % feature_size;

	if (i_feature < embedding_size) {
		// embedding
		int i_x = x[i_batch];
		int i_embedding = i_x * embedding_size + i_feature;
		xl[i] = x_embedder_W[i_embedding];
	}
	else {
		// local
		int i_local = i_batch * local_size + (i_feature - embedding_size);
		xl[i] = l[i_local];
	}
}

__global__ void gruElementWise(
	float* hidden,
	float* W,
	float* U,
	int batch_size,
	int hidden_size
) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * hidden_size) return;

	float W_r_x = W[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float U_r_h = U[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float r = tanh((W_r_x + U_r_h) * 0.5f) * 0.5f + 0.5f;

	float W_z_x = W[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float U_z_h = U[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float z = tanh((W_z_x + U_z_h) * 0.5f) * 0.5f + 0.5f;

	float W_x = W[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float U_x = U[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float h_bar = tanh(W_x + r * U_x);

	hidden[i] = z * hidden[i] + (1.f - z) * h_bar;
}


__global__ void relu(float *x, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	if (x[i] < 0) x[i] = 0;
}


__global__ void initRandomState(hiprandState *state, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	hiprand_init(i, 0, 0, &state[i]);
}


__global__ void addGumbel(float *x, hiprandState *state, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	x[i] += -log(-log(hiprand_uniform(&state[i])));
}


__global__ void pairToKey(int *x, hipcub::KeyValuePair<int, float>* pair, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	x[i] = pair[i].key;
}


void cudaErrorCheckUtil(hipError_t error) {
	if (error != hipSuccess)
	{
		throw sprintf(
			"[Error] %s (error code: %d) at %s line %d\n",
			hipGetErrorString(error),
			error,
			__FILE__,
			__LINE__
		);
	}
}


const char *cublasGetErrorString(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}


void cublasErrorCheckUtil(hipblasStatus_t error) {
	if (error != HIPBLAS_STATUS_SUCCESS)
	{
		throw sprintf(
			"[Error] %s (error code: %d) at %s line %d\n",
			cublasGetErrorString(error),
			error,
			__FILE__,
			__LINE__
		);
	}
}


void cudnnErrorCheckUtil(hipdnnStatus_t error) {
	if (error != HIPBLAS_STATUS_SUCCESS)
	{
		throw sprintf(
			"[Error] %s (error code: %d) at %s line %d\n",
			hipdnnGetErrorString(error),
			error,
			__FILE__,
			__LINE__
		);
	}
}


void inference(
	int batch_size,
	int length,
	int local_size,
	int hidden_size,
	int embedding_size,
	int linear_hidden_size,
	int output_size,
	int* h_output,
	int* h_x,
	float* h_l_array,
	float* h_hidden,
	float* h_x_embedder_W,
	float* h_gru_xw,
	float* h_gru_xb,
	float* h_gru_hw,
	float* h_gru_hb,
	float* h_O1_W,
	float* h_O1_b,
	float* h_O2_W,
	float* h_O2_b
)
{
	// initialize
	std::cout << "initialize" << std::endl;
	auto x = ndarray<int>(batch_size, h_x);
	auto l_array = ndarray<float>(length, batch_size, local_size, h_l_array);
	auto hidden = ndarray<float>(batch_size, hidden_size, h_hidden);

	auto x_embedder_W = ndarray<float>(output_size, embedding_size, h_x_embedder_W);
	auto gru_xw = ndarray<float>(embedding_size + local_size, hidden_size * 3, h_gru_xw);
	auto gru_xb = ndarray<float>(hidden_size * 3, h_gru_xb);
	auto gru_hw = ndarray<float>(hidden_size, hidden_size * 3, h_gru_hw);
	auto gru_hb = ndarray<float>(hidden_size * 3, h_gru_hb);
	auto O1_W = ndarray<float>(hidden_size, linear_hidden_size, h_O1_W);
	auto O1_b = ndarray<float>(linear_hidden_size, h_O1_b);
	auto O2_W = ndarray<float>(linear_hidden_size, output_size, h_O2_W);
	auto O2_b = ndarray<float>(output_size, h_O2_b);

	auto gru_xb_b = ndarray<float>(batch_size, gru_xb.shape1);
	auto gru_hb_b = ndarray<float>(batch_size, gru_hb.shape1);
	auto O1_b_b = ndarray<float>(batch_size, O1_b.shape1);
	auto O2_b_b = ndarray<float>(batch_size, O2_b.shape1);

	auto xl = ndarray<float>(batch_size, embedding_size + local_size);
	auto w_gru_x = ndarray<float>(batch_size, hidden_size * 3);
	auto w_gru_h = ndarray<float>(batch_size, hidden_size * 3);
	auto w_out_x1 = ndarray<float>(batch_size, linear_hidden_size);
	auto w_out_x2 = ndarray<float>(batch_size, output_size);
	auto w_sampled = ndarray<hipcub::KeyValuePair<int, float>>(batch_size);

	auto gumbel_random_state = ndarray<hiprandState>(batch_size, output_size);

	int* h_pinned_output;
	cudaErrorCheckUtil(hipHostAlloc(&h_pinned_output, length  * batch_size * sizeof(int), hipHostMallocDefault));

	// create context
	std::cout << "create context" << std::endl;

	hipStream_t stream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	hipStream_t outputCopyStream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&outputCopyStream, hipStreamNonBlocking));

	hipEvent_t canToKey;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&canToKey, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventRecord(canToKey));

	hipEvent_t canOutputCopy;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&canOutputCopy, hipEventDisableTiming));

	hipblasHandle_t cublasHandle;
	cublasErrorCheckUtil(hipblasCreate(&cublasHandle));
	cublasErrorCheckUtil(hipblasSetStream(cublasHandle, stream));

	for (int i = 0; i < batch_size; i++) {
		// broadcast
		cudaErrorCheckUtil(hipMemcpyAsync(&gru_xb_b.device[i * gru_xb_b.shape2], gru_xb.device, gru_xb_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&gru_hb_b.device[i * gru_hb_b.shape2], gru_hb.device, gru_hb_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&O1_b_b.device[i * O1_b_b.shape2], O1_b.device, O1_b_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&O2_b_b.device[i * O2_b_b.shape2], O2_b.device, O2_b_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
	}

	hipdnnHandle_t cudnnHandle;
	cudnnErrorCheckUtil(hipdnnCreate(&cudnnHandle));
	cudnnErrorCheckUtil(hipdnnSetStream(cudnnHandle, stream));

	hipdnnTensorDescriptor_t softmaxDesc;
	cudnnErrorCheckUtil(hipdnnCreateTensorDescriptor(&softmaxDesc));
	cudnnErrorCheckUtil(hipdnnSetTensor4dDescriptor(
		softmaxDesc,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT,
		w_out_x2.shape1,
		w_out_x2.shape2,
		1,
		1
	));
	initRandomState KERNEL_ARGS4(dim3(512), dim3(gumbel_random_state.size() / 512 + 1), 0, stream) (
		gumbel_random_state.device,  // hiprandState *state,
		gumbel_random_state.size()  // int size
		);

	int* h_argmax_offset = (int*)malloc((w_out_x2.shape1 + 1) * sizeof(int));
	for (int i = 0; i < w_out_x2.shape1 + 1; i++) {
		h_argmax_offset[i] = i * w_out_x2.shape2;
	}
	auto argmax_offset = ndarray<int>(w_out_x2.shape1 + 1, h_argmax_offset);

	size_t argmax_storage_bytes = 0;
	cudaErrorCheckUtil(hipcub::DeviceSegmentedReduce::ArgMax(
		NULL,  // void *d_temp_storage
		argmax_storage_bytes,  // size_t &temp_storage_bytes
		w_out_x2.device,  // InputIteratorT d_in
		w_sampled.device,  // OutputIteratorT d_out
		w_out_x2.shape1,  // int num_segments
		argmax_offset.device,  // OffsetIteratorT d_begin_offsets
		argmax_offset.device + 1,  // OffsetIteratorT d_end_offsets
		0,  // hipStream_t stream
		true  // bool debug_synchronous
	));
	auto argmax_storage = ndarray<char>((int)argmax_storage_bytes);

	std::chrono::system_clock::time_point start, end;
	start = std::chrono::system_clock::now();
	for (int i_local = 0; i_local < length; i_local++) {
		// concat
		concat KERNEL_ARGS4(dim3(512), dim3(xl.size() / 512 + 1), 0, stream) (
			xl.device, // float* xl,
			x.device, // int* x,
			&l_array.device[i_local * (l_array.shape2 * l_array.shape3)], // float* l,
			x_embedder_W.device, // float* x_embedder_W,
			batch_size, // int batch_size,
			local_size, // int local_size,
			embedding_size // int embedding_size
			);

		// gru_x = prev_xl.dot(gru_xw) + gru_xb
		cudaErrorCheckUtil(hipMemcpyAsync(w_gru_x.device, gru_xb_b.device, w_gru_x.size() * sizeof(float), hipMemcpyDeviceToDevice, stream));
		float gemmAlpha = 1, gemmBeta = 1;
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			gru_xw.shape2, // int m,
			xl.shape1, // int n,
			gru_xw.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			gru_xw.device, // const float *A,
			gru_xw.shape2, // int lda,
			xl.device, // const float *B,
			xl.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_gru_x.device, // float *C,
			w_gru_x.shape2 // int ldc
		));

		// gru_h = hidden.dot(gru_hw) + gru_hb
		cudaErrorCheckUtil(hipMemcpyAsync(w_gru_h.device, gru_hb_b.device, w_gru_h.size() * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			gru_hw.shape2, // int m,
			hidden.shape1, // int n,
			gru_hw.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			gru_hw.device, // const float *A,
			gru_hw.shape2, // int lda,
			hidden.device, // const float *B,
			hidden.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_gru_h.device, // float *C,
			w_gru_h.shape2 // int ldc
		));

		// gruElementWise
		gruElementWise KERNEL_ARGS4(dim3(512), dim3(hidden.size() / 512 + 1), 0, stream) (
			hidden.device,  // float* hidden
			w_gru_x.device,  // float* W
			w_gru_h.device,  // float* U
			batch_size,  // int batch_size
			hidden_size  // int hidden_size
			);

		// out_x = hidden.dot(O1_W) + O1_b
		cudaErrorCheckUtil(hipMemcpyAsync(w_out_x1.device, O1_b_b.device, w_out_x1.size() * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			O1_W.shape2, // int m,
			hidden.shape1, // int n,
			O1_W.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			O1_W.device, // const float *A,
			O1_W.shape2, // int lda,
			hidden.device, // const float *B,
			hidden.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_out_x1.device, // float *C,
			w_out_x1.shape2 // int ldc
		));

		// relu
		relu KERNEL_ARGS4(dim3(512), dim3(w_out_x1.size() / 512 + 1), 0, stream) (
			w_out_x1.device,  // float* x
			w_out_x1.size()  // int size
			);

		// out_x = out_x.dot(O2_W) + O2_b
		cudaErrorCheckUtil(hipMemcpyAsync(w_out_x2.device, O2_b_b.device, w_out_x2.size() * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			O2_W.shape2, // int m,
			w_out_x1.shape1, // int n,
			O2_W.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			O2_W.device, // const float *A,
			O2_W.shape2, // int lda,
			w_out_x1.device, // const float *B,
			w_out_x1.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_out_x2.device, // float *C,
			w_out_x2.shape2 // int ldc
		));

		// softmax
		auto dist = w_out_x2;
		float softmaxAlpha = 1, softmaxBeta = 0;
		cudnnErrorCheckUtil(hipdnnSoftmaxForward(
			cudnnHandle, // hipdnnHandle_t
			HIPDNN_SOFTMAX_LOG, // hipdnnSoftmaxAlgorithm_t
			HIPDNN_SOFTMAX_MODE_CHANNEL, // hipdnnSoftmaxMode_t
			&softmaxAlpha, // const void
			softmaxDesc, // const hipdnnTensorDescriptor_t
			dist.device, // const void
			&softmaxBeta, // const void
			softmaxDesc, // const hipdnnTensorDescriptor_t
			dist.device // void
		));

		// sampling
		addGumbel KERNEL_ARGS4(dim3(512), dim3(dist.size() / 512 + 1), 0, stream) (
			dist.device,  // float *x
			gumbel_random_state.device,  // hiprandState *state
			dist.size()  // int size
			);

		cudaErrorCheckUtil(hipcub::DeviceSegmentedReduce::ArgMax(
			argmax_storage.device,  // void *d_temp_storage
			argmax_storage_bytes,  // size_t &temp_storage_bytes
			dist.device,  // InputIteratorT d_in
			w_sampled.device,  // OutputIteratorT d_out
			dist.shape1,  // int num_segments
			argmax_offset.device,  // OffsetIteratorT d_begin_offsets
			argmax_offset.device + 1,  // OffsetIteratorT d_end_offsets
			stream,  // hipStream_t stream
			false  // bool debug_synchronous
		));

		hipStreamWaitEvent(stream, canToKey, 0);
		pairToKey KERNEL_ARGS4(dim3(512), dim3(x.size() / 512 + 1), 0, stream) (
			x.device,  // int *x
			w_sampled.device,  // hipcub::KeyValuePair<int, float>* pair
			x.size()  // int size
			);
		hipEventRecord(canOutputCopy, stream);

		hipStreamWaitEvent(outputCopyStream, canOutputCopy, 0);
		hipMemcpyAsync(&h_pinned_output[i_local * batch_size], x.device, x.size() * sizeof(int), hipMemcpyDeviceToHost, outputCopyStream);
		hipEventRecord(canToKey, outputCopyStream);
	}

	hipStreamSynchronize(stream);
	hipStreamSynchronize(outputCopyStream);
	end = std::chrono::system_clock::now();

	double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000 / 1000;
	printf("time %lf[s]\n", time);

	hipMemcpy(h_output, h_pinned_output, length * batch_size * sizeof(int), hipMemcpyHostToHost);
}

//void main() {
//	inference(NULL);
//}
