#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand/hiprand_kernel.h>
#include <cub.cuh>
#include <util_type.cuh>
#include <>

#include "CppWaveRNN.h"


#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


template<typename T>
T* cudaMallocUtil(int size, T* h = NULL) {
	T* x;
	cudaErrorCheckUtil(hipMalloc(&x, size * sizeof(T)));
	if (h != NULL) {
		cudaErrorCheckUtil(hipMemcpy(x, h, size * sizeof(T), hipMemcpyHostToDevice));
	}
	return x;
}


template<typename T>
struct ndarray {
	T* device;
	T* host;
	int shape1 = 1;
	int shape2 = 1;
	int shape3 = 1;
	ndarray(T* h = NULL) : host(h) {
	}
	ndarray(int s1, T* h = NULL) : shape1(s1), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, T* h = NULL) : shape1(s1), shape2(s2), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}
	ndarray(int s1, int s2, int s3, T* h = NULL) : shape1(s1), shape2(s2), shape3(s3), host(h) {
		device = cudaMallocUtil<T>(size(), host);
	}

	int size() {
		return shape1 * shape2 * shape3;
	}
};


__global__ void concat(float* xl, int* x, float* l, float* x_embedder_W, int batch_size, int local_size, int embedding_size)
{
	int feature_size = embedding_size + local_size;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * feature_size) { return; }

	int i_batch = i / feature_size;
	int i_feature = i % feature_size;

	if (i_feature < embedding_size) {
		// embedding
		int i_x = x[i_batch];
		int i_embedding = i_x * embedding_size + i_feature;
		xl[i] = x_embedder_W[i_embedding];
	}
	else {
		// local
		int i_local = i_batch * local_size + (i_feature - embedding_size);
		xl[i] = l[i_local];
	}
}

__global__ void gruElementWise(
	float* hidden,
	float* W,
	float* U,
	int batch_size,
	int hidden_size
) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= batch_size * hidden_size) return;

	float W_r_x = W[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float U_r_h = U[i % hidden_size + hidden_size * 0 + hidden_size * 3 * (i / hidden_size)];
	float r = tanh((W_r_x + U_r_h) * 0.5f) * 0.5f + 0.5f;

	float W_z_x = W[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float U_z_h = U[i % hidden_size + hidden_size * 1 + hidden_size * 3 * (i / hidden_size)];
	float z = tanh((W_z_x + U_z_h) * 0.5f) * 0.5f + 0.5f;

	float W_x = W[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float U_x = U[i % hidden_size + hidden_size * 2 + hidden_size * 3 * (i / hidden_size)];
	float h_bar = tanh(W_x + r * U_x);

	hidden[i] = z * hidden[i] + (1.f - z) * h_bar;
}


__global__ void relu(float *x, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	if (x[i] < 0) x[i] = 0;
}


__global__ void initRandomState(hiprandState *state, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	hiprand_init(i, 0, 0, &state[i]);
}


__global__ void addGumbel(float *x, hiprandState *state, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	x[i] += -log(-log(hiprand_uniform(&state[i])));
}


__global__ void pairToKey(int *x, hipcub::KeyValuePair<int, float>* pair, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	x[i] = pair[i].key;
}


void cudaErrorCheckUtil(hipError_t error) {
	if (error != hipSuccess)
	{
		std::cout << "[Error] " << hipGetErrorString(error) << "(error code: " << error << ")" << std::endl;
		throw;
	}
}


const char *cublasGetErrorString(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}


void cublasErrorCheckUtil(hipblasStatus_t error) {
	if (error != HIPBLAS_STATUS_SUCCESS)
	{
		std::cout << "[Error] " << cublasGetErrorString(error) << "(error code: " << error << ")" << std::endl;
		throw;
	}
}


void cudnnErrorCheckUtil(hipdnnStatus_t error) {
	if (error != HIPBLAS_STATUS_SUCCESS)
	{
		std::cout << "[Error] " << hipdnnGetErrorString(error) << "(error code: " << error << ")" << std::endl;
		throw;
	}
}

auto g_x = ndarray<int>();
auto g_l_array = ndarray<float>();
auto g_hidden = ndarray<float>();

int* g_h_pinned_output;

hipStream_t g_stream;

hipGraphExec_t g_graphExec;

int g_graph_length;

void initialize(
	int graph_length,
	int batch_size,
	int local_size,
	int hidden_size,
	int embedding_size,
	int linear_hidden_size,
	int output_size,
	float* h_x_embedder_W,
	float* h_gru_xw,
	float* h_gru_xb,
	float* h_gru_hw,
	float* h_gru_hb,
	float* h_O1_W,
	float* h_O1_b,
	float* h_O2_W,
	float* h_O2_b
)
{
	// initialize
	std::cout << "initialize" << std::endl;
	int* h_pinned_x;
	cudaErrorCheckUtil(hipHostAlloc(&h_pinned_x, batch_size * sizeof(int), hipHostMallocDefault));

	float* h_pinned_l_array;
	cudaErrorCheckUtil(hipHostAlloc(&h_pinned_l_array, graph_length  * batch_size * local_size * sizeof(float), hipHostMallocDefault));

	float* h_pinned_hidden;
	cudaErrorCheckUtil(hipHostAlloc(&h_pinned_hidden, batch_size * hidden_size * sizeof(float), hipHostMallocDefault));

	cudaErrorCheckUtil(hipHostAlloc(&g_h_pinned_output, graph_length  * batch_size * sizeof(int), hipHostMallocDefault));

	auto x = ndarray<int>(batch_size, h_pinned_x);
	auto l_array = ndarray<float>(graph_length, batch_size, local_size, h_pinned_l_array);
	auto hidden = ndarray<float>(batch_size, hidden_size, h_pinned_hidden);

	auto x_embedder_W = ndarray<float>(output_size, embedding_size, h_x_embedder_W);
	auto gru_xw = ndarray<float>(embedding_size + local_size, hidden_size * 3, h_gru_xw);
	auto gru_xb = ndarray<float>(hidden_size * 3, h_gru_xb);
	auto gru_hw = ndarray<float>(hidden_size, hidden_size * 3, h_gru_hw);
	auto gru_hb = ndarray<float>(hidden_size * 3, h_gru_hb);
	auto O1_W = ndarray<float>(hidden_size, linear_hidden_size, h_O1_W);
	auto O1_b = ndarray<float>(linear_hidden_size, h_O1_b);
	auto O2_W = ndarray<float>(linear_hidden_size, output_size, h_O2_W);
	auto O2_b = ndarray<float>(output_size, h_O2_b);

	auto gru_xb_b = ndarray<float>(batch_size, gru_xb.shape1);
	auto gru_hb_b = ndarray<float>(batch_size, gru_hb.shape1);
	auto O1_b_b = ndarray<float>(batch_size, O1_b.shape1);
	auto O2_b_b = ndarray<float>(batch_size, O2_b.shape1);

	auto xl = ndarray<float>(batch_size, embedding_size + local_size);
	auto w_gru_x = ndarray<float>(batch_size, hidden_size * 3);
	auto w_gru_h = ndarray<float>(batch_size, hidden_size * 3);
	auto w_out_x1 = ndarray<float>(batch_size, linear_hidden_size);
	auto w_out_x2 = ndarray<float>(batch_size, output_size);
	auto w_sampled = ndarray<hipcub::KeyValuePair<int, float>>(batch_size);

	auto gumbel_random_state = ndarray<hiprandState>(batch_size, output_size);

	// create context
	std::cout << "create context" << std::endl;

	hipStream_t stream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&stream, hipStreamDefault));

	hipStream_t biasCopyStream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&biasCopyStream, hipStreamDefault));

	hipStream_t hiddenStream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&hiddenStream, hipStreamDefault));

	hipStream_t outputCopyStream;
	cudaErrorCheckUtil(hipStreamCreateWithFlags(&outputCopyStream, hipStreamDefault));

	hipEvent_t elementWiseDone, gemmO2Done, argmaxDone;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&elementWiseDone, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventCreateWithFlags(&gemmO2Done, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventCreateWithFlags(&argmaxDone, hipEventDisableTiming));

	hipEvent_t copyGruXbDone, copyGruHbDone, copyO1bDone, copyO2bDone;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&copyGruXbDone, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventCreateWithFlags(&copyGruHbDone, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventCreateWithFlags(&copyO1bDone, hipEventDisableTiming));
	cudaErrorCheckUtil(hipEventCreateWithFlags(&copyO2bDone, hipEventDisableTiming));

	hipEvent_t gemmO1Done;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&gemmO1Done, hipEventDisableTiming));

	hipEvent_t gemmGruHDone;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&gemmGruHDone, hipEventDisableTiming));

	hipEvent_t outputCopyDone;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&outputCopyDone, hipEventDisableTiming));

	hipEvent_t toKeyDone;
	cudaErrorCheckUtil(hipEventCreateWithFlags(&toKeyDone, hipEventDisableTiming));

	hipblasHandle_t cublasHandle;
	cublasErrorCheckUtil(hipblasCreate(&cublasHandle));
	cublasErrorCheckUtil(hipblasSetStream(cublasHandle, stream));

	hipblasHandle_t cublasHiddenHandle;
	cublasErrorCheckUtil(hipblasCreate(&cublasHiddenHandle));
	cublasErrorCheckUtil(hipblasSetStream(cublasHiddenHandle, hiddenStream));

	for (int i = 0; i < batch_size; i++) {
		// broadcast
		cudaErrorCheckUtil(hipMemcpyAsync(&gru_xb_b.device[i * gru_xb_b.shape2], gru_xb.device, gru_xb_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&gru_hb_b.device[i * gru_hb_b.shape2], gru_hb.device, gru_hb_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&O1_b_b.device[i * O1_b_b.shape2], O1_b.device, O1_b_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
		cudaErrorCheckUtil(hipMemcpyAsync(&O2_b_b.device[i * O2_b_b.shape2], O2_b.device, O2_b_b.shape2 * sizeof(float), hipMemcpyDeviceToDevice, stream));
	}

	hipdnnHandle_t cudnnHandle;
	cudnnErrorCheckUtil(hipdnnCreate(&cudnnHandle));
	cudnnErrorCheckUtil(hipdnnSetStream(cudnnHandle, stream));

	hipdnnTensorDescriptor_t softmaxDesc;
	cudnnErrorCheckUtil(hipdnnCreateTensorDescriptor(&softmaxDesc));
	cudnnErrorCheckUtil(hipdnnSetTensor4dDescriptor(
		softmaxDesc,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_FLOAT,
		w_out_x2.shape1,
		w_out_x2.shape2,
		1,
		1
	));
	initRandomState KERNEL_ARGS4(dim3(512), dim3(gumbel_random_state.size() / 512 + 1), 0, stream) (
		gumbel_random_state.device,  // hiprandState *state,
		gumbel_random_state.size()  // int size
		);

	int* h_argmax_offset = (int*)malloc((w_out_x2.shape1 + 1) * sizeof(int));
	for (int i = 0; i < w_out_x2.shape1 + 1; i++) {
		h_argmax_offset[i] = i * w_out_x2.shape2;
	}
	auto argmax_offset = ndarray<int>(w_out_x2.shape1 + 1, h_argmax_offset);

	size_t argmax_storage_bytes = 0;
	cudaErrorCheckUtil(hipcub::DeviceSegmentedReduce::ArgMax(
		NULL,  // void *d_temp_storage
		argmax_storage_bytes,  // size_t &temp_storage_bytes
		w_out_x2.device,  // InputIteratorT d_in
		w_sampled.device,  // OutputIteratorT d_out
		w_out_x2.shape1,  // int num_segments
		argmax_offset.device,  // OffsetIteratorT d_begin_offsets
		argmax_offset.device + 1,  // OffsetIteratorT d_end_offsets
		0,  // hipStream_t stream
		true  // bool debug_synchronous
	));
	auto argmax_storage = ndarray<char>((int)argmax_storage_bytes);

	std::cout << "graph start" << std::endl;

	cudaErrorCheckUtil(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
	hipEventRecord(elementWiseDone, stream);  // for joining

	cudaErrorCheckUtil(hipMemcpyAsync(x.device, x.host, x.size() * sizeof(int), hipMemcpyHostToDevice, stream));
	cudaErrorCheckUtil(hipMemcpyAsync(l_array.device, l_array.host, l_array.size() * sizeof(float), hipMemcpyHostToDevice, stream));
	cudaErrorCheckUtil(hipMemcpyAsync(hidden.device, hidden.host, hidden.size() * sizeof(float), hipMemcpyHostToDevice, stream));

	for (int i_local = 0; i_local < graph_length; i_local++) {
		// concat
		concat KERNEL_ARGS4(dim3(512), dim3(xl.size() / 512 + 1), 0, stream) (
			xl.device, // float* xl,
			x.device, // int* x,
			&l_array.device[i_local * (l_array.shape2 * l_array.shape3)], // float* l,
			x_embedder_W.device, // float* x_embedder_W,
			batch_size, // int batch_size,
			local_size, // int local_size,
			embedding_size // int embedding_size
			);

		// gru_x = prev_xl.dot(gru_xw) + gru_xb
		hipStreamWaitEvent(biasCopyStream, elementWiseDone, 0);
		cudaErrorCheckUtil(hipMemcpyAsync(w_gru_x.device, gru_xb_b.device, w_gru_x.size() * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
		hipEventRecord(copyGruXbDone, biasCopyStream);

		float gemmAlpha = 1, gemmBeta = 1;
		hipStreamWaitEvent(stream, copyGruXbDone, 0);
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			gru_xw.shape2, // int m,
			xl.shape1, // int n,
			gru_xw.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			gru_xw.device, // const float *A,
			gru_xw.shape2, // int lda,
			xl.device, // const float *B,
			xl.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_gru_x.device, // float *C,
			w_gru_x.shape2 // int ldc
		));

		// gru_h = hidden.dot(gru_hw) + gru_hb
		hipStreamWaitEvent(biasCopyStream, elementWiseDone, 0);
		cudaErrorCheckUtil(hipMemcpyAsync(w_gru_h.device, gru_hb_b.device, w_gru_h.size() * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
		hipEventRecord(copyGruHbDone, biasCopyStream);

		hipStreamWaitEvent(hiddenStream, copyGruHbDone, 0);
		hipStreamWaitEvent(hiddenStream, gemmO1Done, 0);
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHiddenHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			gru_hw.shape2, // int m,
			hidden.shape1, // int n,
			gru_hw.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			gru_hw.device, // const float *A,
			gru_hw.shape2, // int lda,
			hidden.device, // const float *B,
			hidden.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_gru_h.device, // float *C,
			w_gru_h.shape2 // int ldc
		));
		hipEventRecord(gemmGruHDone, hiddenStream);

		// gruElementWise
		hipStreamWaitEvent(stream, gemmGruHDone, 0);
		gruElementWise KERNEL_ARGS4(dim3(512), dim3(hidden.size() / 512 + 1), 0, stream) (
			hidden.device,  // float* hidden
			w_gru_x.device,  // float* W
			w_gru_h.device,  // float* U
			batch_size,  // int batch_size
			hidden_size  // int hidden_size
			);
		hipEventRecord(elementWiseDone, stream);

		// out_x = hidden.dot(O1_W) + O1_b
		hipStreamWaitEvent(biasCopyStream, gemmO2Done, 0);
		cudaErrorCheckUtil(hipMemcpyAsync(w_out_x1.device, O1_b_b.device, w_out_x1.size() * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
		hipEventRecord(copyO1bDone, biasCopyStream);

		hipStreamWaitEvent(stream, copyO1bDone, 0);
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			O1_W.shape2, // int m,
			hidden.shape1, // int n,
			O1_W.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			O1_W.device, // const float *A,
			O1_W.shape2, // int lda,
			hidden.device, // const float *B,
			hidden.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_out_x1.device, // float *C,
			w_out_x1.shape2 // int ldc
		));
		hipEventRecord(gemmO1Done, stream);

		// relu
		relu KERNEL_ARGS4(dim3(512), dim3(w_out_x1.size() / 512 + 1), 0, stream) (
			w_out_x1.device,  // float* x
			w_out_x1.size()  // int size
			);

		// out_x = out_x.dot(O2_W) + O2_b
		hipStreamWaitEvent(biasCopyStream, argmaxDone, 0);
		cudaErrorCheckUtil(hipMemcpyAsync(w_out_x2.device, O2_b_b.device, w_out_x2.size() * sizeof(float), hipMemcpyDeviceToDevice, biasCopyStream));
		hipEventRecord(copyO2bDone, biasCopyStream);

		hipStreamWaitEvent(stream, copyO2bDone, 0);
		cublasErrorCheckUtil(hipblasSgemm(
			cublasHandle, // hipblasHandle_t handle,
			HIPBLAS_OP_N, // hipblasOperation_t transa,
			HIPBLAS_OP_N, // hipblasOperation_t transb,
			O2_W.shape2, // int m,
			w_out_x1.shape1, // int n,
			O2_W.shape1, // int k,
			&gemmAlpha, // const float *alpha, /* host or device pointer */
			O2_W.device, // const float *A,
			O2_W.shape2, // int lda,
			w_out_x1.device, // const float *B,
			w_out_x1.shape2, // int ldb,
			&gemmBeta, // const float *beta, /* host or device pointer */
			w_out_x2.device, // float *C,
			w_out_x2.shape2 // int ldc
		));
		hipEventRecord(gemmO2Done, stream);

		// softmax
		auto dist = w_out_x2;
		float softmaxAlpha = 1, softmaxBeta = 0;
		cudnnErrorCheckUtil(hipdnnSoftmaxForward(
			cudnnHandle, // hipdnnHandle_t
			HIPDNN_SOFTMAX_LOG, // hipdnnSoftmaxAlgorithm_t
			HIPDNN_SOFTMAX_MODE_CHANNEL, // hipdnnSoftmaxMode_t
			&softmaxAlpha, // const void
			softmaxDesc, // const hipdnnTensorDescriptor_t
			dist.device, // const void
			&softmaxBeta, // const void
			softmaxDesc, // const hipdnnTensorDescriptor_t
			dist.device // void
		));

		// sampling
		addGumbel KERNEL_ARGS4(dim3(512), dim3(dist.size() / 512 + 1), 0, stream) (
			dist.device,  // float *x
			gumbel_random_state.device,  // hiprandState *state
			dist.size()  // int size
			);

		cudaErrorCheckUtil(hipcub::DeviceSegmentedReduce::ArgMax(
			argmax_storage.device,  // void *d_temp_storage
			argmax_storage_bytes,  // size_t &temp_storage_bytes
			dist.device,  // InputIteratorT d_in
			w_sampled.device,  // OutputIteratorT d_out
			dist.shape1,  // int num_segments
			argmax_offset.device,  // OffsetIteratorT d_begin_offsets
			argmax_offset.device + 1,  // OffsetIteratorT d_end_offsets
			stream,  // hipStream_t stream
			false  // bool debug_synchronous
		));
		hipEventRecord(argmaxDone, stream);

		hipStreamWaitEvent(stream, outputCopyDone, 0);
		pairToKey KERNEL_ARGS4(dim3(512), dim3(x.size() / 512 + 1), 0, stream) (
			x.device,  // int *x
			w_sampled.device,  // hipcub::KeyValuePair<int, float>* pair
			x.size()  // int size
			);
		hipEventRecord(toKeyDone, stream);

		hipStreamWaitEvent(outputCopyStream, toKeyDone, 0);
		hipMemcpyAsync(&g_h_pinned_output[i_local * batch_size], x.device, x.size() * sizeof(int), hipMemcpyDeviceToHost, outputCopyStream);
		hipEventRecord(outputCopyDone, outputCopyStream);
	}

	hipMemcpyAsync(x.host, x.device, x.size() * sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(hidden.host, hidden.device, hidden.size() * sizeof(float), hipMemcpyDeviceToHost, stream);

	hipStreamWaitEvent(stream, outputCopyDone, 0);

	hipGraph_t graph;
	cudaErrorCheckUtil(hipStreamEndCapture(stream, &graph));
	std::cout << "graph done" << std::endl;

	cudaErrorCheckUtil(hipGraphInstantiate(&g_graphExec, graph, NULL, NULL, 0));

	// destroy
	cudaErrorCheckUtil(hipStreamDestroy(biasCopyStream));
	cudaErrorCheckUtil(hipStreamDestroy(hiddenStream));
	cudaErrorCheckUtil(hipStreamDestroy(outputCopyStream));

	// global parameters
	g_x = x;
	g_l_array = l_array;
	g_hidden = hidden;

	g_stream = stream;

	g_graph_length = graph_length;
}


void inference(
	int batch_size,
	int length,
	int* h_output,
	int* h_x,
	float* h_l_array,
	float* h_hidden
)
{
	// launch
	std::chrono::system_clock::time_point start, end;
	start = std::chrono::system_clock::now();

	cudaErrorCheckUtil(hipMemcpy(g_x.host, h_x, g_x.size() * sizeof(int), hipMemcpyHostToHost));
	cudaErrorCheckUtil(hipMemcpy(g_hidden.host, h_hidden, g_hidden.size() * sizeof(float), hipMemcpyHostToHost));

	for (int i_loop = 0; i_loop < length / g_graph_length; i_loop++) {
		hipMemcpy(g_l_array.host, &h_l_array[i_loop * g_l_array.size()], g_l_array.size() * sizeof(float), hipMemcpyHostToHost);

		cudaErrorCheckUtil(hipGraphLaunch(g_graphExec, g_stream));

		hipMemcpy(&h_output[i_loop * g_graph_length * batch_size], g_h_pinned_output, g_graph_length * batch_size * sizeof(int), hipMemcpyHostToHost);
	}

	cudaErrorCheckUtil(hipMemcpy(h_x, g_x.host, g_x.size() * sizeof(int), hipMemcpyHostToHost));
	cudaErrorCheckUtil(hipMemcpy(h_hidden, g_hidden.host, g_hidden.size() * sizeof(float), hipMemcpyHostToHost));

	end = std::chrono::system_clock::now();

	double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()) / 1000 / 1000;
	printf("time %lf[s]\n", time);
}
